#include "hip/hip_runtime.h"
/*
 dynamics.cu
*/
#include "../include/dynamics.cuh"

__host__ __device__ float Cart_type_Pendulum_ddx(float u, float x, float theta, float dx, float dtheta, SystemControlVariable *SCV)
{
    float a[10];
    /*a[0] = state[3] + powf(state[2], 2) * state[1];		//J+l^2*mp
	a[1] = u - dx * state[4]
			+ powf(dtheta, 2) * state[2] * state[1] * sinf(theta);//u-dx*myuc+dtheta^2*l*mp*sin
	a[2] = cosf(theta) * state[2] * state[1];						//cos*l*mp
	a[3] = dtheta * state[5] - state[6] * state[2] * state[1] * sinf(theta);//dtheta*myup-g*l*mp*sin
	a[4] = -(a[0] * a[1] + a[2] * a[3]);

	a[5] = powf(cosf(theta), 2) * powf(state[2], 2) * powf(state[1], 2);//cos^2*l^2*mp^2
	a[6] = state[0] + state[1];		//mc+mp
	a[7] = state[3] + powf(state[2], 2) * state[1];		//J+l^2*mp
	a[8] = a[5] - (a[6] * a[7]);*/

    a[0] = SCV->params[3] + powf(SCV->params[2], 2) * SCV->params[1]; // J + mplp^2
    a[1] = u - dx * SCV->params[4] + powf(dtheta, 2) * SCV->params[2] * SCV->params[1] * sinf(theta); // u - dx * myu_c *dtheta^2 * mp * lp * sin(theta)
    a[2] = cosf(theta) * SCV->params[2] * SCV->params[1]; // mp * lp * cos(theta)
    a[3] = dtheta * SCV->params[5] - SCV->params[6] * SCV->params[2] * SCV->params[1] * sinf(theta); //myu_p * dtheta - mp * g * lp * sin(theta)
    a[4] = -(a[0] * a[1] + a[2] * a[3]);

    a[5] = powf(cosf(theta), 2) * powf(SCV->params[2], 2) * powf(SCV->params[1], 2); //mp^2 * lp^2 * cos^2(theta)
    a[6] = SCV->params[0] + SCV->params[1]; // mp + mc
    a[7] = SCV->params[3] + powf(SCV->params[2],2) * SCV->params[1];
    a[8] = a[5] - (a[6] * a[7]);

	return a[4] / a[8];
}

__host__ __device__ float Cart_type_Pendulum_ddtheta(float u, float x,  float theta, float dx, float dtheta, SystemControlVariable *SCV)
{
    float a[10];
    /*a[0] = cosf(theta) * state[2] * state[1];		//cos*l*mp
	a[1] = u - dx * state[4]
			+ powf(dtheta, 2) * state[2] * state[1] * sinf(theta);//u-dx*myuc+dtheta^2*l*mp*sin
	a[2] = state[0] + state[1];		//mc+mp
	a[3] = dtheta * state[5] - state[6] * state[2] * state[1] * sinf(theta);//dtheta*myup-g*l*mp*sin
	a[4] = -(a[0] * a[1] + a[2] * a[3]);

	a[5] = state[3] * (state[0] + state[1]);		//J(mc+mp)
	a[6] = powf(state[2], 2) * state[1];		//l^2*mp
	a[7] = state[0] + state[1] - powf(cosf(theta), 2) * state[1];//mc+mp-cos^2*mp
	a[8] = a[5] + a[6] * a[7];*/

    a[0] = cosf(theta) * SCV->params[2] * SCV->params[1]; //mp * lp * cos(theta)
    a[1] = u - dx * SCV->params[4] + powf(dtheta, 2) * SCV->params[2] * SCV->params[1] * sinf(theta); //u - myu_c * dx + mp * lp * dtheta^2 *sin(theta)
    a[2] = SCV->params[0] + SCV->params[1]; //mc + mp
    a[3] = dtheta * SCV->params[5] - SCV->params[6] * SCV->params[2] * SCV->params[1] * sinf(theta); // myu_p * dtheta - mp * lp * sin(theta)
    a[4] = -(a[0] * a[1] + a[2] * a[3]);

    a[5] = SCV->params[3] * (SCV->params[0] + SCV->params[1]);
    a[6] = powf(SCV->params[2], 2) * SCV->params[1];
    a[7] = SCV->params[0] + SCV->params[1] - powf(cosf(theta), 2) * SCV->params[1];
    a[8] = a[5] + a[6] * a[7];

	return a[4] / a[8];
}

__host__ __device__ void get_Lx_Cart_and_SinglePole(float *Lx, Tolerance *prev, SystemControlVariable *SCV)
{
    // float temp_Lx[DIM_OF_STATES] = { };
    Lx[0] = SCV->weightMatrix[0] * prev->state[0];
    Lx[1] = SCV->weightMatrix[1] * cosf(prev->state[1] / 2) * sinf(prev->state[1] / 2) * 0.5f;
    Lx[2] = SCV->weightMatrix[2] * prev->state[2];
    Lx[3] = SCV->weightMatrix[3] * prev->state[3];
}

__host__ __device__ void get_LFx_Cart_and_SinglePole(float *LFx, Tolerance *current, Tolerance *later, SystemControlVariable *SCV, float t_delta)
{
    float a[36] = { };
    float /*x,*/ th, dx, dtheta;
    // x = current->state[0];
    th = current->state[1];
    dx = current->state[2];
    dtheta = current->state[3];

    a[0] = - later->lambda[0]; //lambda^T * Fx(:,1) := LFx[0]

    a[1] = -powf(SCV->params[1] * SCV->params[2] * cosf(th), 2) + powf(SCV->params[1] * SCV->params[2], 2)
            + SCV->params[0] * SCV->params[1] * powf(SCV->params[2], 2) + SCV->params[3] * (SCV->params[0] + SCV->params[1]); // -(Mp*lp*cos(th))^2 + (Mp*lp)^2 + Mc*Mp*lp^2 + Jp * (Mp + Mc)
    a[2] = (SCV->params[1] * powf(SCV->params[2], 2) + SCV->params[3]) * SCV->params[1] * powf(dtheta, 2) * SCV->params[2] * cosf(th); // (Jp + Mp*lp^2) * Mp * lp * dth^2 * cos(th)
    a[3] = powf(SCV->params[1] * SCV->params[2] * cosf(th), 2) * SCV->params[6]; //(Mp*lp*cos(th))^2 * g
    a[4] = (SCV->params[5] * dtheta - SCV->params[1] * SCV->params[6] * SCV->params[2] * sinf(th)); // (mup * dtheta - Mp * lp * g * sin(th) )
    a[5] = SCV->params[1] * SCV->params[2] * sinf(th); //Mp * lp * sin(th)
    a[6] = a[4] * a[5]; //Mp * lp * sin(th) *(mup * dtheta - Mp * lp * g * sin(th) )
    a[7] = 2.0f * powf(SCV->params[1] * SCV->params[2], 3) * powf(cosf(th), 2) * sinf(th); //2 * Mp^3 * lp^3 * cos^2(th) * sin(th)
    a[8] = 2.0f * powf(SCV->params[1] * SCV->params[2], 2) * cosf(th) * sinf(th); //2 * Mp^2 * lp^2 * cos(th) * sin(th)
    a[9] = SCV->params[3] + SCV->params[1] * powf(SCV->params[2], 2); // Jp + Mp * lp^2
    a[10] = SCV->params[1] * SCV->params[2] * powf(dtheta, 2) * sinf(th) + current->Input[0] - SCV->params[4] * dx; // [Mp * lp * dtheta^2 * sinf(th) + U - muc * dx]
    a[11] = SCV->params[1] * SCV->params[2]; //Mp * lp
    a[12] = 2.0f * SCV->params[1] * powf( SCV->params[2], 2) * dtheta * sinf(th) + SCV->params[5] * cosf(th) + 2.0f * SCV->params[3] * sinf(th); //2 * Mp * lp^2 *dtheta * sin(th) + mup * cos(th) + 2 * Jp * dtheta * sin(th)
    
    a[13] = a[2] / a[1]; //[Fx;32]_1
    a[14] = a[3] / a[1]; //[Fx;32]_2
    a[15] = a[6] / a[1]; //[Fx;32]_3
    a[16] = a[1] * a[1]; //a[1]^2
    a[17] = (a[7] * a[4]) / a[16]; //[Fx;32]_4
    a[18] = (a[8] * a[9] * a[10]) / a[16]; //[Fx;32]_5
    a[19] = (a[13] - a[14] - a[15] - a[17] - a[18]) * t_delta; // Fx;32
    
    a[20] = -(SCV->params[4] * a[9]) / a[1]; //[Fx;33] = -myuc * (Jp + Mp * lp^2) / a[1]
    a[21] = a[20] * t_delta - 1.0f; // Fx;33

    a[22] = SCV->params[1] * SCV->params[2] * a[12] / a[1]; //[Fx;34] = Mp * lp * (2 * Mp * lp^2 * dtheta * sin(th) + myup * cos(th) + 2 * Jp * dtheta * sinf(th) ) / a[1]
    a[23] = a[22] * t_delta; //Fx;34

    a[24] = (a[5] * a[10]) / a[1]; //[Fx;42]_1
    a[25] =  powf(SCV->params[1] * SCV->params[2] * cosf(th), 2); // (Mp*lp*cos(th))^2
    a[26] = a[22] * powf(dtheta, 2) / a[1]; //[Fx;42]_2
    a[27] = a[11] * SCV->params[6] * cosf(th) * (SCV->params[0] + SCV->params[1]) / a[1]; //[Fx;42]_3
    a[28] = (a[7] * a[10]) / a[16]; //[Fx;42]_4
    a[29] = (a[8] * (SCV->params[0] + SCV->params[1]) * a[4]) / a[16]; //[Fx;42]_5
    a[30] = (a[24] - a[26] + a[27] + a[28] + a[29]) * t_delta; // Fx;42 

    a[31] = SCV->params[5] * SCV->params[1] * SCV->params[2] * cosf(th) / a[1]; //[Fx;43]
    a[32] = a[31] * t_delta; //Fx;43

    a[33] = -SCV->params[5] * (SCV->params[0] + SCV->params[1]) / a[1]; //[Fx;44]_1
    a[34] = -a[8] / a[1]; //[Fx;44]_2
    a[35] = (a[29] + a[30]) * t_delta - 1.0f; //Fx;44

    // 行列Fxの面倒臭い要素の計算まで”一応終了”<--- 2021.07.12

    LFx[0] = a[0];
    LFx[1] = -later->lambda[1] + a[19] * later->lambda[2] + a[30] * later->lambda[3];
    LFx[2] = (t_delta * later->lambda[0]) + (a[21] * later->lambda[2]) + (a[32] * later->lambda[3]);
    LFx[3] = (t_delta * later->lambda[1]) + (a[23] * later->lambda[2]) + (a[35] * later->lambda[3]); 
}

__host__ __device__ void get_dHdu_Cart_and_SinglePole(Tolerance *current, Tolerance *later, SystemControlVariable *SCV, float t_delta)
{
    float temp_Lu[DIM_OF_INPUT] = { };
    float temp_LBu[DIM_OF_INPUT] = { };
    float temp_Fu[DIM_OF_STATES] = { };
    float temp_LamFu[DIM_OF_INPUT] = { };

    float o[10] = { };
    temp_Lu[0] = SCV->weightMatrix[4] * current->Input[0];
    
    o[0] = powf(current->Input[0], 2) - powf(SCV->constraints[1], 2); // (U^2 - U_max^2)
    o[1] = 2.0f * current->Input[0]; // 2 * U
    o[2] = o[1] / o[0]; //LBu = 2U / (U^2 - U_max^2)

    o[3] = SCV->params[3] + SCV->params[1] * powf(SCV->params[2], 2); //Jp + Mp * lp^2
    o[4] = -SCV->params[1] * SCV->params[2] * cosf(current->state[1]); // -Mp * lp * cos(th)
    
    o[5] = -powf(SCV->params[1] * SCV->params[2] * cosf(current->state[1]), 2); //- Mp^2 * lp^2 * cos(th)^2
    o[6] = powf(SCV->params[1] * SCV->params[2], 2); // Mp^2 * lp^2
    o[7] = SCV->params[0] * SCV->params[1] * powf(SCV->params[2], 2); // Mc * Mp * lp^2
    o[8] = SCV->params[3] * (SCV->params[0] + SCV->params[1]); // Jp * (Mp + Mc)
    o[9] = o[5] + o[6] + o[7] + o[8];

    temp_LBu[0] = o[2];

    temp_Fu[2] = (o[3] * t_delta) / o[9];
    temp_Fu[3] = (o[4] * t_delta) / o[9];

    for(int i = 0; i < DIM_OF_INPUT; i++)
    {
        for(int k = 0; k < DIM_OF_STATES; k++)
        {
            temp_LamFu[i] += later->lambda[k] * temp_Fu[k];
        }
    }

    for(int i = 0; i < DIM_OF_INPUT; i++)
    {
        current->dHdu[i] = temp_Lu[i] + Rho * temp_LBu[i] + temp_LamFu[i];
    }
}

void get_curent_diff_state_Cart_and_SinglePole(float *diffState, float *state, float input, SystemControlVariable *SCV)
{
    diffState[0] = state[2]; //dx
    diffState[1] = state[3]; //dtheta
    diffState[2] = Cart_type_Pendulum_ddx(input, state[0], state[1], state[2], state[3], SCV);
    diffState[3] = Cart_type_Pendulum_ddtheta(input, state[0], state[1], state[2], state[3], SCV);
}

void Eular_integrator(float *yp_vector, float t_delta, float *diffState)
{
    for(int i = 0; i < DIM_OF_STATES; i++)
    {
        yp_vector[i] = diffState[i] * t_delta;
    }
}

void Runge_Kutta45_for_SecondaryOderSystem(SystemControlVariable *SCV, float input, float t_delta)
{
    float state[DIM_OF_STATES], diff_state[DIM_OF_STATES], yp_1[DIM_OF_STATES], next_state[DIM_OF_STATES];
    // float params[DIM_OF_PARAMETERS] = { };
    // copyStateFromDataStructure(state, SCV);
    for(int i = 0; i < DIM_OF_STATES; i++)
    {
        state[i] = SCV->state[i];
    }
    /*for(int i = 0; i < DIM_OF_PARAMETERS; i++){
        params[i] = SCV->params[i];
    }*/
    get_curent_diff_state_Cart_and_SinglePole( diff_state, state, input, SCV);
    Eular_integrator(yp_1, t_delta, diff_state);
    for(int i = 0; i < DIM_OF_STATES; i++)
    {
        next_state[i] = state[i] + yp_1[i] / 2;
    }

    float yp_2[DIM_OF_STATES] = { };
    get_curent_diff_state_Cart_and_SinglePole( diff_state, next_state, input, SCV);
    Eular_integrator(yp_2, t_delta, diff_state);
    for(int i = 0; i < DIM_OF_STATES; i++)
    {
        next_state[i] = state[i] + yp_2[i] / 2;
    }

    float yp_3[DIM_OF_STATES] = { };
    get_curent_diff_state_Cart_and_SinglePole( diff_state, next_state, input, SCV);
    Eular_integrator(yp_3, t_delta, diff_state);
    for(int i = 0; i < DIM_OF_STATES; i++)
    {
        next_state[i] = state[i] + yp_3[i];
    }

    float yp_4[DIM_OF_STATES] = { };
    get_curent_diff_state_Cart_and_SinglePole( diff_state, next_state, input, SCV);
    Eular_integrator(yp_4, t_delta, diff_state);

    for(int i = 0; i < DIM_OF_STATES; i++)
    {
        SCV->state[i] = state[i] + (yp_1[i] + 2 * yp_2[i] + 2 * yp_3[i] + yp_4[i]) / 6.0f;
    }
}